#include "hip/hip_runtime.h"
/**
* Prototype for Parallel Matrix Multiplication Algorithm
**/

#include <iostream>
#include <math.h>
#include "DenseMatrix.cpp"


using namespace std;

__global__
void FGEMM(int n, DenseMatrix<float> *a, DenseMatrix<float> *b, DenseMatrix<float> *c) {

    // Will store calculated results from each thread 
    extern __shared__ float calculatedResults[]; 

    if (a->GetRows() == b->GetColumns()) {

    	// Every threadblock gets at least 1 row of matrix A and 1 column of matrix B
    	// There could be an exception to this if matrices are particularly large
		// Each thread gets 1 value from the row and 1 value from the column
		// Then it does the operation on those values

		// Gets value from row of DenseMatrix A 
    	float rowVal =  (*a)(blockIdx.x, threadIdx.x);

    	// Gets value from column of DenseMatrix B
    	float colVal = (*b)(threadIdx.x, blockIdx.x);


    	// Perform thread operation
    	calculatedResults[threadIdx.x] = rowVal * colVal;


    	// Perform reduction on calculated values to add them up.
    	// Resulting value will be assigned to location in DenseMatrix c
    	// This can be made much faster by doing a parallel reduction but this can be added later
    	
    	__syncthreads();

    	if (threadIdx.x == 1) {
	    	for (int i = 1; i < sizeof(calculatedResults)/sizeof(calculatedResults[0]); i++) {
	    			calculatedResults[0] += calculatedResults[i];
	    		}

    		(*c)(blockIdx.x, blockIdx.x) = calculatedResults[0];
     	}
    }

}



int main() {

	int N = 6;
    
    DenseMatrix<float>
        oDM1{{1,2,3},{4,5,6}},
        oDM2{{4,9},{12347,835},{91,7532}};


    hipMalloc(&oDM1, sizeof(DenseMatrix<float>));
    hipMalloc(&oDM2, sizeof(DenseMatrix<float>));


    //FGEMM<<<1, 1>>>(N, oDM1, oDM2);


}

