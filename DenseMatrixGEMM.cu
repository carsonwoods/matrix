#include "hip/hip_runtime.h"
/**
* Prototype for Parallel Matrix Multiplication Algorithm
**/

#include <iostream>
#include <math.h>
#include "DenseMatrix.cpp"


using namespace std;

__global__
void FGEMM(int n, DenseMatrix<float> *a, DenseMatrix<float> *b) {

        

}



int main() {

	int N = 6;
    
    DenseMatrix<float>
        oDM1{{1,2,3},{4,5,6}},
        oDM2{{4,9},{12347,835},{91,7532}};


    hipMalloc((void **)&oDM1, sizeof(DenseMatrix<float>));t
    hipMalloc((void **)&oDM2, sizeof(DenseMatrix<float>));


    FGEMM<<<1, 1>>>(N, oDM1, oDM2)

    cout << "Hello World" << endl;

}

